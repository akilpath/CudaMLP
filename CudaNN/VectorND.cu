#include "hip/hip_runtime.h"
#include "VectorND.cuh"



__global__ void _add_vector(float* out, float* data_1, float* data_2, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (index < size) {
		out[index] = data_1[index] + data_2[index];
		index += stride;
	}
}

__global__ void _subtract_vector(float* out, float* data_1, float* data_2, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (index < size) {
		out[index] = data_1[index] - data_2[index];
		index += stride;
	}
}

__global__ void _multiply_vector(float* out, float* data_1, float* data_2, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (index < size) {
		out[index] = data_1[index] * data_2[index];
		index += stride;
	}
}

__global__ void _vec_scalar_multiply(float* out, float* data, int size, float scalar) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		out[index] = data[index] * scalar;
	}
}

VectorND::VectorND(int size) {
	this->size_ = size;
	hipMallocManaged(&data_, size * sizeof(float));
	for (int i = 0; i < size; i++) {
		this->data_[i] = 0;
	}
}

VectorND::VectorND(int size, float* in_data)
{
	this->size_ = size;
	hipMallocManaged(&data_, size * sizeof(float));
	for (int i = 0; i < size; i++) {
		this->data_[i] = in_data[i];
	}
}
VectorND::VectorND() {
	this->size_ = 0;
	this->data_ = nullptr;
}

VectorND::~VectorND() {
	if (data_ != nullptr) {
		hipFree(data_);
	}
}

int VectorND::size(){
	return this->size_;
}


void VectorND::print_data() const {
	std::cout << "[";
	for (int i = 0; i < this->size_; i++) {
		if (i == this->size_ - 1)
			std::cout << this->data_[i];
		else
			std::cout << this->data_[i] << ", ";
	}
	std::cout << "]" << std::endl;
}

int VectorND::scalar_multiply(VectorND &out, float scalar)
{
	if (out.size() != this->size()) {
		printf("Error in VectorND::scalar_multiply: Output vector size does not match input vector size\n");
		return -1;
	}
	const int block_size = 128;
	int num_blocks = (int)ceil(this->size() / block_size);
	if (num_blocks == 0) {
		num_blocks = 1;
	}
	_vec_scalar_multiply << <num_blocks, block_size >> > (out.data_, data_, size_, scalar);
	hipDeviceSynchronize();
	return 0;
}

int VectorND::vector_add(VectorND& out, VectorND& in)
{
	if (size_ != in.size()) {
		std::cout << "Error: Vector sizes do not match\n";
		return -1;
	}

	if (out.size() != size_) {
		std::cout << "Error: Output vector size does not match input vector size\n";
		return -1;
	}

	int block_size = 128;
	int num_blocks = (int)ceil(this->size() / block_size);
	if (num_blocks == 0) {
		num_blocks = 1;
	}
	_add_vector << <num_blocks, block_size >> > (out.data_, this->data_, in.data_, size_);
	hipDeviceSynchronize();
	return 0;
}

int VectorND::vector_subtract(VectorND& out, VectorND& in)
{
	if (out.size() != in.size()) {
		return -1;
	}
	if (out.size() != this->size()) {
		return -1;
	}

	const int block_size = 128;
	int num_blocks = (int)ceil(this->size() / block_size);
	_subtract_vector << <num_blocks, block_size >> > (out.data_, this->data_, in.data_, this->size());
	hipDeviceSynchronize();
	return 0;
}

int VectorND::vector_multiply(VectorND& out, VectorND& in)
{
	if (out.size() != in.size()) {
		return -1;
	}
	if (out.size() != this->size()) {
		return -1;
	}

	const int block_size = 128;
	int num_blocks = (int)ceil(this->size() / block_size);
	_multiply_vector << <num_blocks, block_size >> > (out.data_, this->data_, in.data_, this->size());
	hipDeviceSynchronize();
	return 0;
}




